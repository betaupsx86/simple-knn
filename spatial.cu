#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"
#include <hip/hip_runtime.h>

torch::Tensor
distCUDA2(const torch::Tensor& points)
{
  hipSetDevice(points.device().index());
  const int P = points.size(0);

  torch::Tensor means = torch::full({P}, 0.0, points.options().dtype(torch::kFloat32));  
  
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

  return means;
}